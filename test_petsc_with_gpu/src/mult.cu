#include "hip/hip_runtime.h"
#include "mult.cuh"

__device__ void compute_jacobian_basis(
    const int in_p, const int in_q, const double h1, const double h2,
    const double *d_B1, const double *d_B2,
    const double *d_dB1, const double *d_dB2,
    const double *s_nurbs_extraction1, const double *s_nurbs_extraction2,
    const double *eCP,
    double &jacobian,
    double *R)
{
    const int p = 3;
    const int q = 3;
    double N1[p + 1];
    double dN1[p + 1];
    for (int i = 0; i < p + 1; ++i)
    {
        N1[i] = 0.0;
        dN1[i] = 0.0;
    }
    double N2[q + 1];
    double dN2[q + 1];
    for (int j = 0; j < q + 1; ++j)
    {
        N2[j] = 0.0;
        dN2[j] = 0.0;
    }

    for (int jj = 0; jj < p + 1; ++jj)
    {
        for (int kk = 0; kk < p + 1; ++kk)
        {
            N1[jj] += s_nurbs_extraction1[jj * (p + 1) + kk] * d_B1[kk];
            dN1[jj] += s_nurbs_extraction1[jj * (p + 1) + kk] * d_dB1[kk];
        }
        dN1[jj] /= h1;
    }
    for (int jj = 0; jj < q + 1; ++jj)
    {
        for (int kk = 0; kk < q + 1; ++kk)
        {
            N2[jj] += s_nurbs_extraction2[jj * (q + 1) + kk] * d_B2[kk];
            dN2[jj] += s_nurbs_extraction2[jj * (q + 1) + kk] * d_dB2[kk];
        }
        dN2[jj] /= h2;
    }

    const int nLocBas = (p + 1) * (q + 1);
    double N[nLocBas];
    double dN_dxi[nLocBas];
    double dN_deta[nLocBas];
    double w = 0.0;
    double dw_dxi = 0.0;
    double dw_deta = 0.0;
    for (int j = 0; j < q + 1; ++j)
    {
        for (int i = 0; i < p + 1; ++i)
        {
            N[j * (p + 1) + i] = N1[i] * N2[j];
            w += N[j * (p + 1) + i];
            dN_dxi[j * (p + 1) + i] = dN1[i] * N2[j];
            dw_dxi += dN_dxi[j * (p + 1) + i];
            dN_deta[j * (p + 1) + i] = N1[i] * dN2[j];
            dw_deta += dN_deta[j * (p + 1) + i];
        }
    }

    double dR_dxi[nLocBas];
    double dR_deta[nLocBas];
    for (int j = 0; j < q + 1; ++j)
    {
        for (int i = 0; i < p + 1; ++i)
        {
            R[j * (p + 1) + i] = N[j * (p + 1) + i] / w;
            dR_dxi[j * (p + 1) + i] = (dN_dxi[j * (p + 1) + i] - dw_dxi * R[j * (p + 1) + i]) / w;
            dR_deta[j * (p + 1) + i] = (dN_deta[j * (p + 1) + i] - dw_deta * R[j * (p + 1) + i]) / w;
        }
    }

    double dx_dxi = 0.0;
    double dx_deta = 0.0;
    double dy_dxi = 0.0;
    double dy_deta = 0.0;

    for (int j = 0; j < q + 1; ++j)
    {
        for (int i = 0; i < p + 1; ++i)
        {
            dx_dxi += eCP[2 * (j * (p + 1) + i)] * dR_dxi[j * (p + 1) + i];
            dx_deta += eCP[2 * (j * (p + 1) + i)] * dR_deta[j * (p + 1) + i];
            dy_dxi += eCP[2 * (j * (p + 1) + i) + 1] * dR_dxi[j * (p + 1) + i];
            dy_deta += eCP[2 * (j * (p + 1) + i) + 1] * dR_deta[j * (p + 1) + i];
        }
    }
    jacobian = dx_dxi * dy_deta - dx_deta * dy_dxi;
    jacobian *= h1*h2;
}

__device__ void compute_jacobian_derivative(
    const int in_p, const int in_q, const double h1, const double h2,
    const double *d_B1, const double *d_B2,
    const double *d_dB1, const double *d_dB2,
    const double *s_nurbs_extraction1, const double *s_nurbs_extraction2,
    const double *eCP, 
    double &jacobian,
    double *dR_dx, 
    double *dR_dy)
{
    const int p = 3;
    const int q = 3;
    double N1[p + 1];
    double dN1[p + 1];
    for (int i = 0; i < p + 1; ++i)
    {
        N1[i] = 0.0;
        dN1[i] = 0.0;
    }
    double N2[q + 1];
    double dN2[q + 1];
    for (int j = 0; j < q + 1; ++j)
    {
        N2[j] = 0.0;
        dN2[j] = 0.0;
    }

    for (int jj = 0; jj < p + 1; ++jj)
    {
        for (int kk = 0; kk < p + 1; ++kk)
        {
            N1[jj] +=  s_nurbs_extraction1[jj * (p + 1) + kk] * d_B1[kk];
            dN1[jj] += s_nurbs_extraction1[jj * (p + 1) + kk] * d_dB1[kk];
        }
        dN1[jj] /= h1;
    }
    for (int jj = 0; jj < q + 1; ++jj)
    {
        for (int kk = 0; kk < q + 1; ++kk)
        {
            N2[jj] += s_nurbs_extraction2[jj * (q + 1) + kk] * d_B2[kk];
            dN2[jj] += s_nurbs_extraction2[jj * (q + 1) + kk] * d_dB2[kk];
        }
        dN2[jj] /= h2;
    }

    const int nLocBas = (p + 1) * (q + 1);
    double N[nLocBas];
    double dN_dxi[nLocBas];
    double dN_deta[nLocBas];
    double w = 0.0;
    double dw_dxi = 0.0;
    double dw_deta = 0.0;

    for (int j = 0; j < q + 1; ++j)
    {
        for (int i = 0; i < p + 1; ++i)
        {
            N[j * (p + 1) + i] = N1[i] * N2[j];
            w += N[j * (p + 1) + i];
            dN_dxi[j * (p + 1) + i] = dN1[i] * N2[j];
            dw_dxi += dN_dxi[j * (p + 1) + i];
            dN_deta[j * (p + 1) + i] = N1[i] * dN2[j];
            dw_deta += dN_deta[j * (p + 1) + i];
        }
    }

    double R[nLocBas];
    double dR_dxi[nLocBas];
    double dR_deta[nLocBas];

    for (int j = 0; j < q + 1; ++j)
    {
        for (int i = 0; i < p + 1; ++i)
        {
            R[j * (p + 1) + i] = N[j * (p + 1) + i] / w;
            dR_dxi[j * (p + 1) + i] = (dN_dxi[j * (p + 1) + i] - dw_dxi * R[j * (p + 1) + i]) / w;
            dR_deta[j * (p + 1) + i] = (dN_deta[j * (p + 1) + i] - dw_deta * R[j * (p + 1) + i]) / w;
        }
    }

    double dx_dxi = 0.0;
    double dx_deta = 0.0;
    double dy_dxi = 0.0;
    double dy_deta = 0.0;
    double dxi_dx = 0.0;
    double dxi_dy = 0.0;
    double deta_dx = 0.0;
    double deta_dy = 0.0;

    for (int j = 0; j < q + 1; ++j)
    {
        for (int i = 0; i < p + 1; ++i)
        {
            dx_dxi += eCP[2 * (j * (p + 1) + i)] * dR_dxi[j * (p + 1) + i];
            dx_deta += eCP[2 * (j * (p + 1) + i)] * dR_deta[j * (p + 1) + i];
            dy_dxi += eCP[2 * (j * (p + 1) + i) + 1] * dR_dxi[j * (p + 1) + i];
            dy_deta += eCP[2 * (j * (p + 1) + i) + 1] * dR_deta[j * (p + 1) + i];
        }
    }

    jacobian = dx_dxi * dy_deta - dx_deta * dy_dxi;

    dxi_dx = dy_deta / jacobian;
    dxi_dy = -dx_deta / jacobian;
    deta_dx = -dy_dxi / jacobian;
    deta_dy = dx_dxi / jacobian;

    for (int j = 0; j < q + 1; ++j)
    {
        for (int i = 0; i < p + 1; ++i)
        {
            dR_dx[j * (p + 1) + i] = dxi_dx * dR_dxi[j * (p + 1) + i] + deta_dx * dR_deta[j * (p + 1) + i];
            dR_dy[j * (p + 1) + i] = dxi_dy * dR_dxi[j * (p + 1) + i] + deta_dy * dR_deta[j * (p + 1) + i];
        }
    }

    jacobian *= h1 * h2;
}

__device__ double get_force(double x, double y)
{
    return x * (1.0 - x) * y * (1.0 - y);
}

__global__ void AssembleKernel(const int in_p, const int in_q,
    double *d_B1, double *d_B2,
    double *d_dB1, double *d_dB2,
    double *d_nurbs_extraction1, double *d_nurbs_extraction2,
    double *d_elem_size1, double *d_elem_size2,
    int *d_IEN, int *d_ID,
    double *d_CP,
    double *qw1, double *qw2,
    double *d_x_array
    )
{
    const int p = 3;
    const int q = 3;
    extern __shared__ char shared_data[];

    int offset = 0;
    int *s_eID = (int*)(shared_data + offset);
    offset += (p + 1) * (q + 1) * sizeof(int);
    double *s_eCP = (double*)(shared_data + offset);
    offset += 2 * (p + 1) * (q + 1) * sizeof(double);
    double *s_eNURBSExtraction1 = (double*)(shared_data + offset);
    offset += (p + 1) * (p + 1) * sizeof(double);
    double *s_eNURBSExtraction2 = (double*)(shared_data + offset);
    offset += (q + 1) * (q + 1) * sizeof(double);
    double *s_qw = (double*)(shared_data + offset);

    int elemIndex = blockIdx.y * gridDim.x + blockIdx.x;
    const int nLocBas = (p + 1) * (q + 1);

    for (int j = 0; j < nLocBas; ++j)
    {
        s_eID[j] = d_ID[d_IEN[elemIndex * nLocBas + j]];
        s_eCP[2 * j] = d_CP[2 * d_IEN[elemIndex * nLocBas + j]];
        s_eCP[2 * j + 1] = d_CP[2 * d_IEN[elemIndex * nLocBas + j] + 1];
    }

    for (int j = 0; j < q + 1; ++j)
    {
        for (int i = 0; i < p + 1; ++i)
        {
            s_qw[j * (p + 1) + i] = qw1[i] * qw2[j];
        }
    }

    for (int i = 0; i < (p + 1) * (p + 1); ++i)
        s_eNURBSExtraction1[i] = d_nurbs_extraction1[blockIdx.x * (p + 1) * (p + 1) + i];
    for (int i = 0; i < (q + 1) * (q + 1); ++i)
        s_eNURBSExtraction2[i] = d_nurbs_extraction2[blockIdx.y * (q + 1) * (q + 1) + i];

    double h1 = d_elem_size1[blockIdx.x];
    double h2 = d_elem_size2[blockIdx.y];

    __syncthreads();

    int qpx = threadIdx.x;
    int qpy = threadIdx.y;
    int qp = threadIdx.y * blockDim.x + threadIdx.x;

    double B1[p + 1];
    double dB1[p + 1];
    double B2[q + 1];
    double dB2[q + 1];

    if (qp < (p+1)*(q+1))
    {
        for (int i = 0; i < p + 1; ++i)
        {
            B1[i] = d_B1[qpx * (p + 1) + i];
            dB1[i] = d_dB1[qpx * (p + 1) + i];
        }
        for (int j = 0; j < q + 1; ++j)
        {
            B2[j] = d_B2[qpy * (q + 1) + j];
            dB2[j] = d_dB2[qpy * (q + 1) + j];
        }

        double jacobian;
        double R[nLocBas];

        compute_jacobian_basis(p, q, h1, h2, B1, B2, dB1, dB2,
            s_eNURBSExtraction1, s_eNURBSExtraction2, s_eCP, jacobian, R);
        
        double x = 0.0;
        double y = 0.0;

        for (int ii = 0; ii < nLocBas; ++ii)
        {
            x += s_eCP[2 * ii] * R[ii];
            y += s_eCP[2 * ii + 1] * R[ii];
        }

        double force = get_force(x, y);

        for (int ii = 0; ii < nLocBas; ++ii)
        {
            double val = R[ii] * force * jacobian * s_qw[qp];
            int coo_index = s_eID[ii];
            if (coo_index >= 0)
            {
                atomicAdd(&d_x_array[coo_index], val);
            }
        }
    }
}

__global__ void MatrixFreeMatMultKernel(const int in_p, const int in_q,
    double *d_B1, double *d_B2,
    double *d_dB1, double *d_dB2,
    double *d_nurbs_extraction1, double *d_nurbs_extraction2,
    double *d_elem_size1, double *d_elem_size2,
    int *d_IEN, int *d_ID,
    double *d_CP,
    double *qw1, double *qw2,
    const double *d_F_array_in,
    double *d_F_array_out
    )
{
    extern __shared__ char shared_data[];

    const int p = 3;
    const int q = 3;
    const int nLocBas = (p + 1) * (q + 1);

    int offset = 0;
    int *s_eID = (int*)(shared_data + offset);
    offset += nLocBas * sizeof(int);
    double *s_eCP = (double*)(shared_data + offset);
    offset += 2 * nLocBas * sizeof(double);
    double *s_eNURBSExtraction1 = (double*)(shared_data + offset);
    offset += (p + 1) * (p + 1) * sizeof(double);
    double *s_eNURBSExtraction2 = (double*)(shared_data + offset);
    offset += (q + 1) * (q + 1) * sizeof(double);
    double *s_qw = (double*)(shared_data + offset);
    offset += (p + 1) * (q + 1) * sizeof(double);
    double *Floc_in = (double*)(shared_data + offset);
    offset += nLocBas * sizeof(double);
    double *Floc_out = (double*)(shared_data + offset);
    
    int elemIndex = blockIdx.y * gridDim.x + blockIdx.x;

    for (int j = 0; j < nLocBas; ++j)
    {
        s_eID[j] = d_ID[d_IEN[elemIndex * nLocBas + j]];
        s_eCP[2 * j] = d_CP[2 * d_IEN[elemIndex * nLocBas + j]];
        s_eCP[2 * j + 1] = d_CP[2 * d_IEN[elemIndex * nLocBas + j] + 1];
    }

    for (int j = 0; j < q + 1; ++j)
    {
        for (int i = 0; i < p + 1; ++i)
        {
            s_qw[j * (p + 1) + i] = qw1[i] * qw2[j];
        }
    }

    for (int i = 0; i < (p + 1) * (p + 1); ++i)
        s_eNURBSExtraction1[i] = d_nurbs_extraction1[blockIdx.x * (p + 1) * (p + 1) + i];
    for (int i = 0; i < (q + 1) * (q + 1); ++i)
        s_eNURBSExtraction2[i] = d_nurbs_extraction2[blockIdx.y * (q + 1) * (q + 1) + i];

    for (int i = 0; i < nLocBas; ++i)
    {
        int coo_index = d_IEN[elemIndex * nLocBas + i];
        Floc_in[i] = d_F_array_in[coo_index];
        Floc_out[i] = 0.0;
    }

    double h1 = d_elem_size1[blockIdx.x];
    double h2 = d_elem_size2[blockIdx.y];

    __syncthreads();

    int qpx = threadIdx.x;
    int qpy = threadIdx.y;
    int qp = threadIdx.y * blockDim.x + threadIdx.x;

    double B1[p + 1];
    double dB1[p + 1];
    double B2[q + 1];
    double dB2[q + 1];

    if (qp < (p+1)*(q+1))
    {
        for (int i = 0; i < p + 1; ++i)
        {
            B1[i] = d_B1[qpx * (p + 1) + i];
            dB1[i] = d_dB1[qpx * (p + 1) + i];
        }
        for (int j = 0; j < q + 1; ++j)
        {
            B2[j] = d_B2[qpy * (q + 1) + j];
            dB2[j] = d_dB2[qpy * (q + 1) + j];
        }

        double jacobian;
        double dR_dx[nLocBas];
        double dR_dy[nLocBas];

        compute_jacobian_derivative(p, q, h1, h2, B1, B2, dB1, dB2,
            s_eNURBSExtraction1, s_eNURBSExtraction2, s_eCP, jacobian, dR_dx, dR_dy);
        
        double temp_x = 0.0;
        double temp_y = 0.0;

        for (int jj = 0; jj < nLocBas; ++jj)
        {
            temp_x += dR_dx[jj] * Floc_in[jj];
            temp_y += dR_dy[jj] * Floc_in[jj];
        }

        temp_x *= -s_qw[qp]*jacobian;
        temp_y *= -s_qw[qp]*jacobian;

        for (int ii = 0; ii < nLocBas; ++ii)
        {
            Floc_out[ii] += (dR_dx[ii] * temp_x + dR_dy[ii] * temp_y);
        }

        for (int ii = 0; ii < nLocBas; ++ii)
        {
            int coo_index = s_eID[ii];
            if (coo_index >= 0)
            {
                atomicAdd(&d_F_array_out[coo_index], Floc_out[ii]);
            }
        }
    }
}

__global__ void DirichletBCKernel(const int * d_Dir, const int dirsize, double * d_val, double value)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < dirsize)
    {
        int coo_index = d_Dir[idx];
        if (coo_index >= 0)
        {
            d_val[coo_index] = value;
        }
    }
}

void AssembleLoadCUDA(const int p, const int q,
    const int nlocalelemx, const int nlocalelemy,
    double * d_B1, double * d_B2,
    double * d_dB1, double * d_dB2,
    double * d_nurbs_extraction1, double * d_nurbs_extraction2,
    double * d_elem_size1, double * d_elem_size2,
    int * d_IEN, int * d_ID, double * d_CP,
    double * qw1, double * qw2, double * d_F_array)
{
    int shared_size = (p + 1) * (q + 1) * sizeof(int)
                + 2 * (p + 1) * (q + 1) * sizeof(double)
                + (p + 1) * (p + 1) * sizeof(double)
                + (q + 1) * (q + 1) * sizeof(double)
                + (p + 1) * (q + 1) * sizeof(double);

    AssembleKernel<<<dim3(nlocalelemx, nlocalelemy), dim3(p+1, q+1), shared_size>>>(
        p, q, d_B1, d_B2, d_dB1, d_dB2,
        d_nurbs_extraction1, d_nurbs_extraction2,
        d_elem_size1, d_elem_size2,
        d_IEN, d_ID, d_CP,
        qw1, qw2, d_F_array);

    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("Error in AssembleLoadCUDA: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

void MatrixFreeMatMultCUDA(const int p, const int q,
    const int nlocalelemx, const int nlocalelemy,
    double * d_B1, double * d_B2,
    double * d_dB1, double * d_dB2,
    double * d_nurbs_extraction1, double * d_nurbs_extraction2,
    double * d_elem_size1, double * d_elem_size2,
    int * d_IEN, int * d_ID, double * d_CP,
    double * qw1, double * qw2,
    const double * d_F_array_in, double * d_F_array_out)
{
    int shared_size = (p + 1) * (q + 1) * sizeof(int)
                + 2 * (p + 1) * (q + 1) * sizeof(double)
                + (p + 1) * (p + 1) * sizeof(double)
                + (q + 1) * (q + 1) * sizeof(double)
                + (p + 1) * (q + 1) * sizeof(double)
                + (p + 1) * (q + 1) * sizeof(double)
                + (p + 1) * (q + 1) * sizeof(double);

    MatrixFreeMatMultKernel<<<dim3(nlocalelemx, nlocalelemy), dim3(p+1, q+1), shared_size>>>(
        p, q, d_B1, d_B2, d_dB1, d_dB2,
        d_nurbs_extraction1, d_nurbs_extraction2,
        d_elem_size1, d_elem_size2,
        d_IEN, d_ID, d_CP,
        qw1, qw2,
        d_F_array_in, d_F_array_out);

    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("Error in MatrixFreeMatMultCUDA: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

void DirichletBCCUDA(const int * d_Dir, const int dirsize, double * d_x_array, double value)
{
    int blocksize = 256;
    int nblocks = (dirsize + blocksize - 1) / blocksize;

    DirichletBCKernel<<<nblocks, blocksize>>>(d_Dir, dirsize, d_x_array, value);

    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("Error in DirichletBCCUDA: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}