#include <hip/hip_runtime.h>
#include "memory.hpp"

void MallocDeviceMemory(void **ptr, size_t size)
{
    hipError_t err = hipMalloc(ptr, size);
    if (err != hipSuccess) {
        fprintf(stderr, "Error allocating device memory: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

void FreeDeviceMemory(void *ptr)
{
    hipError_t err = hipFree(ptr);
    if (err != hipSuccess) {
        fprintf(stderr, "Error freeing device memory: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

void CopyToDevice(void *dst, const void *src, size_t size)
{
    hipError_t err = hipMemcpy(dst, src, size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Error copying to device memory: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}