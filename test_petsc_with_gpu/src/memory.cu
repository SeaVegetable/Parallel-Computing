#include <hip/hip_runtime.h>
#include <stdio.h>
#include "memory.hpp"

template <class T>
void MallocDeviceMemory(T **ptr, size_t n)
{
    hipError_t err = hipMalloc((void**)ptr, n * sizeof(T));
    if (err != hipSuccess) {
        fprintf(stderr, "Error allocating device memory: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

template <class T>
void FreeDeviceMemory(T *ptr)
{
    hipError_t err = hipFree(ptr);
    if (err != hipSuccess) {
        fprintf(stderr, "Error freeing device memory: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

template <class T>
void CopyToDevice(T *dst, const T *src, size_t n)
{
    hipError_t err = hipMemcpy(dst, src, n * sizeof(T), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Error copying to device memory: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}